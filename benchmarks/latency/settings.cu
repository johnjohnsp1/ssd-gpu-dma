#include <hip/hip_runtime.h>
#include "settings.h"
#include <nvm_types.h>
#include <nvm_cmd.h>
#include <nvm_aq.h>
#include <sstream>
#include <string>
#include <cstdint>
#include <getopt.h>

using std::string;

static const struct option options[] = {
    { .name = "help", .has_arg = no_argument, .flag = nullptr, .val = 'h' },
    { .name = "ctrl", .has_arg = required_argument, .flag = nullptr, .val = 'c' },
    { .name = "cuda-device", .has_arg = required_argument, .flag = nullptr, .val = 'g' },
    { .name = "gpu", .has_arg = required_argument, .flag = nullptr, .val = 'g' },
    { .name = "namespace", .has_arg = required_argument, .flag = nullptr, .val = 'i' },
    { .name = "adapter", .has_arg = required_argument, .flag = nullptr, .val = 'a' },
    { .name = "num-blocks", .has_arg = required_argument, .flag = nullptr, .val = 'n' },
    { .name = "blocks", .has_arg = required_argument, .flag = nullptr, .val = 'n' },
    { .name = "offset", .has_arg = required_argument, .flag  = nullptr, .val = 'o' },
    { .name = "queues", .has_arg = required_argument, .flag = nullptr, .val = 'q' },
    { .name = "depth", .has_arg = required_argument, .flag = nullptr, .val = 'd' },
    { .name = "warmups", .has_arg = required_argument, .flag = nullptr, .val = 'w' },
    { .name = "repetitions", .has_arg = required_argument, .flag = nullptr, .val = 'r' },
    { .name = "repeat", .has_arg = no_argument, .flag = nullptr, .val = AccessPattern::REPEAT },
    { .name = "sequential", .has_arg = no_argument, .flag = nullptr, .val =  AccessPattern::SEQUENTIAL },
    { .name = "random", .has_arg = no_argument, .flag = nullptr, .val = AccessPattern::RANDOM },
    { .name = "verify", .has_arg = required_argument, .flag = nullptr, .val = 'v' },
    { .name = nullptr, .has_arg = no_argument, .flag = nullptr, .val = 0 }
};



static string usageString(const char* name)
{
    return name + string(": [-a adapter] --ctrl id [--gpu-no] --num-blocks block-count [-q queue-count] [--depth queue-depth]");
}



static string helpString(const char* name)
{
    string usage(usageString(name));
    return usage;
}


static int maxCudaDevice()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess)
    {
        throw string("Unexpected error: ") + hipGetErrorString(err);
    }
    return deviceCount;
}



Settings::Settings()
{
    cudaDevice = -1;
    controllerId = 0;
    adapter = 0;
    segmentId = 0;
    nvmNamespace = 1;
    warmups = 10;
    repetitions = 1000;
    numQueues = 1;
    queueDepth = 32;
    numBlocks = 0;
    startBlock = 0;
    pattern = SEQUENTIAL;
    filename = nullptr;
}


static uint64_t parseNumber(const char* str, int base)
{
    char* end = nullptr;
    uint64_t n = strtoul(str, &end, base);

    if (end == nullptr || *end != '\0')
    {
        throw string("Invalid number: `") + str + string("'");
    }

    return n;
}


static uint64_t parseNumber(const char* str)
{
    return parseNumber(str, 0);
}



void Settings::parseArguments(int argc, char** argv)
{
    int index;
    int option;

    while ((option = getopt_long(argc, argv, ":hc:g:i:a:n:o:q:d:w:r:v:", options, &index)) != -1)
    {
        switch (option)
        {
            case '?':
                throw string("Unknown option: `") + argv[optind - 1] + string("'");

            case ':':
                throw string("Missing argument for option ") + argv[optind - 1];

            case 'h':
                throw helpString(argv[0]);

            case AccessPattern::REPEAT:
            case AccessPattern::SEQUENTIAL:
            case AccessPattern::RANDOM:
                pattern = AccessPattern(option);
                break;

            case 'c':
                controllerId = (uint64_t) parseNumber(optarg);
                break;

            case 'g':
                cudaDevice = (int) parseNumber(optarg, 10);
                if (cudaDevice < 0 || cudaDevice >= maxCudaDevice())
                {
                    throw string("Invalid CUDA device: ") + optarg;
                }
                break;

            case 'i':
                nvmNamespace = (uint32_t) parseNumber(optarg);
                if (nvmNamespace == NVM_CMD_NS_ALL || nvmNamespace == 0)
                {
                    throw string("Not a valid NVM namespace: ") + optarg;
                }
                break;

            case 'a':
                adapter = (uint32_t) parseNumber(optarg, 10);
                if (adapter >= NVM_DIS_RPC_MAX_ADAPTER)
                {
                    throw string("Invalid adapter number: ") + optarg;
                }
                break;

            case 'n':
                numBlocks = (size_t) parseNumber(optarg);
                if (numBlocks == 0)
                {
                    throw string("Number of blocks must be at least 1");
                }
                break;

            case 'o':
                startBlock = (size_t) parseNumber(optarg);
                break;

            case 'q':
                numQueues = (size_t) parseNumber(optarg);
                if (numQueues == 0 || numQueues > 0xffff)
                {
                    throw string("Invalid number of IO queues specified, must be in range 1-65535");
                }
                break;

            case 'd':
                queueDepth = (size_t) parseNumber(optarg);
                if (queueDepth < 1 || queueDepth >= 64)
                {
                    throw string("Invalid queue depth, must be in range 1-64");
                }
                break;

            case 'w':
                warmups = (size_t) parseNumber(optarg);
                break;

            case 'r':
                repetitions = (size_t) parseNumber(optarg);
                break;

            case 'v':
                filename = optarg;
                break;
        }
    }

    if (controllerId == 0)
    {
        throw string("No controller specified!");
    }

    if (numBlocks == 0)
    {
        throw string("No length is specified!");
    }
}

